#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>




using namespace std;

// New
#define gpuErrCheck( err ) (gpuAssert( err, __FILE__, __LINE__ ))

static void gpuAssert(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		#ifdef _WIN32
			system("pause");
		#endif
		exit(EXIT_FAILURE);
	}
}

int main()
{
    cout << "Hello World\n" << endl;

    int n = 10000; 
    int *a;
    int *dev_a;

	if (NULL == (a = (int*)malloc(n*sizeof(int)))) exit(20);

	int c = 32 + 23;
	c += 1;

	printf("%d\n", c);


	gpuErrCheck(hipMalloc((void**)&dev_a, n * sizeof(int)));
	gpuErrCheck(hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice));

}